#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "test.h"

// these macros are really really helpful
#  define CUDA_SAFE_CALL( call) {                                            \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

#define CHECKLASTERROR   {                                                 \
	hipError_t err = hipGetLastError();                                    \
	if (err != hipSuccess) {                                                \
		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
	} }


#define SIZ 128

__global__ void truc(unsigned int * buf) {
	if (threadIdx.x < SIZ) {
		buf[threadIdx.x] = buf[threadIdx.x] << 5;
	}
	__syncthreads();
}

int testcuda()
{
	unsigned int* foo = (unsigned int*) malloc(SIZ * sizeof(unsigned int));
	for (int x = 0; x < SIZ; ++x) {
		foo[x] = 1;
	}

	unsigned int * recf = NULL;
	CUDA_SAFE_CALL( hipMalloc((void **) &recf, SIZ * sizeof(unsigned int)) );
	CUDA_SAFE_CALL(hipMemcpy(recf, foo,  SIZ * sizeof(unsigned int), hipMemcpyHostToDevice));
	truc<<<1, SIZ>>>(recf);
	CHECKLASTERROR
	CUDA_SAFE_CALL(hipMemcpy(foo, recf, SIZ * sizeof(unsigned int), hipMemcpyDeviceToHost));
	printf("2^5 -> %u\n", foo[5]);

	CUDA_SAFE_CALL(hipFree(recf));

/*
int deviceCount;
hipGetDeviceCount(&deviceCount);
printf("device count %d\n", deviceCount);

hipDeviceProp_t dP;
hipGetDeviceProperties(&dP, NULL);
//printf("Max threads per block: %d\n", dP.maxThreadsPerBlock);
//printf("Max Threads DIM: %d x %d x %d\n", dP.maxThreadsDim[0], dP.maxThreadsDim[1], dP.maxThreadsDim[2]);
//printf("Max Grid Size: %d x %d x %d\n", dP.maxGridSize[0], dP.maxGridSize[1], dP.maxGridSize[2]);

hipDeviceProp_t* pDeviceProp = &dP;

     printf( "\nDevice Name \t - %s ", pDeviceProp->name );  
     printf( "\n**************************************");  
     printf( "\nTotal Global Memory\t\t -%d KB", pDeviceProp->totalGlobalMem/1024 );  
     printf( "\nShared memory available per block \t - %d KB", pDeviceProp->sharedMemPerBlock/1024 );  
     printf( "\nNumber of registers per thread block \t - %d", pDeviceProp->regsPerBlock );  
     printf( "\nWarp size in threads \t - %d", pDeviceProp->warpSize );  
     printf( "\nMemory Pitch \t - %d bytes", pDeviceProp->memPitch );  
     printf( "\nMaximum threads per block \t - %d", pDeviceProp->maxThreadsPerBlock );  
     printf( "\nMaximum Thread Dimension (block) \t - %d %d %d", pDeviceProp->maxThreadsDim[0], pDeviceProp->maxThreadsDim[1], pDeviceProp->maxThreadsDim[2] );  
     printf( "\nMaximum Thread Dimension (grid) \t - %d %d %d", pDeviceProp->maxGridSize[0], pDeviceProp->maxGridSize[1], pDeviceProp->maxGridSize[2] );  
     printf( "\nTotal constant memory \t - %d bytes", pDeviceProp->totalConstMem );  
     printf( "\nCUDA ver \t - %d.%d", pDeviceProp->major, pDeviceProp->minor );  
     printf( "\nClock rate \t - %d KHz", pDeviceProp->clockRate );  
     printf( "\nTexture Alignment \t - %d bytes", pDeviceProp->textureAlignment );  
     printf( "\nDevice Overlap \t - %s", pDeviceProp-> deviceOverlap?"Allowed":"Not Allowed" );  
     printf( "\nNumber of Multi processors \t - %d\n", pDeviceProp->multiProcessorCount );  
*/

	return 0;
}

